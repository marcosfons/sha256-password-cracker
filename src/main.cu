#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <stddef.h>
#include <sys/time.h>
#include <pthread.h>
#include <locale.h>
#include <math.h>

#include "sha256.cuh"
#include "hash_entry.cuh"
#include "cuda_devices.cuh"


// Current stage
// #define DEBUG
#define TEST
// #define RELEASE

#define TEST_TYPE SEQUENTIALLY
// #define TEST_TYPE RANDOMLY

#define GPUS 1

#ifdef DEBUG
#define BLOCKS 1
#define THREADS 1
#define RUNS_PER_ITERATION 1
#define PRINT_STATUS_DELAY 10000
#endif

#ifdef TEST
#define BLOCKS 32
#define BLOCKS_PER_ENTRY 16
#define THREADS 1024
#define RUNS_PER_ITERATION 8
#define LOOPS_INSIDE_THREAD 64
#define PRINT_STATUS_DELAY 50000
#endif

#ifdef RELEASE
#define BLOCKS 16
#define THREADS 2048
#define RUNS_PER_ITERATION 128
#define PRINT_STATUS_DELAY 1000000
#endif

#define THREAD_EXECUTION_ITERATIONS ((MAX_PASSWORD_LENGTH - MIN_PASSWORD_CHECK))

__constant__ const char charset[] = {
    'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n',
    'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z', 'A', 'B',
    'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P',
    'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z', '1', '2', '3', '4',
    '5', '6', '7', '8', '9', '0', '%', '*', '$', '@'};

__constant__ const int CHARSET_LENGTH = sizeof(charset) / sizeof(char);

typedef struct handler_input {
  int device;
  unsigned long long hashesProcessed;
	unsigned long long start;
	hash_entry* entries;
	int entries_count;
	unsigned char finished;
} handler_input;

long long timems() {
	struct timeval end;
	gettimeofday(&end, NULL);
	return (long long) end.tv_sec * 1000 + (long long) end.tv_usec / 1000;
}

__device__ unsigned long deviceRandomGen(unsigned long x) {
	x ^= (x << 21);
	x ^= (x >> 35);
	x ^= (x << 4);
	return x;
}

__device__ unsigned char hash_cmp_equal(const unsigned char hash1[HASH_BYTES_LENGTH], const unsigned char hash2[HASH_BYTES_LENGTH]) {
	// if (hash1[0] != hash2[0]) {
	// 	return 0;
	// }
	#pragma unroll
	for (int i = 0; i < HASH_BYTES_LENGTH; i++) {
		if (hash1[i] != hash2[i]) {
			return 0;
		}
	}
	return 1;
	// return 1;
	// return (hash1.hash_number_long[0]   == hash2.hash_number_long[0] ) &&
	// 				(hash1.hash_number_long[1]  == hash2.hash_number_long[1] ) &&
	// 				(hash1.hash_number[1]       == hash2.hash_number[1]) &&
	// 				(hash1.hash_number[2]       == hash2.hash_number[2]) &&
	// 				(hash1.hash_number[3]       == hash2.hash_number[3]);
}

__device__
int get_input_from_number(unsigned long long current, unsigned char input[8]) {
	#pragma unroll
	for (int i = 0; i < 8; i++) {
		input[i] = charset[current % CHARSET_LENGTH];
		current /= CHARSET_LENGTH;
		if (current <= 0) {
			return i + 1;
		}
	}
	return 0;
}

__global__ void sha256_cuda_all_posibilities(hash_entry *entries, int entries_count,
                             unsigned char *blockContainsSolution,
                             unsigned long long start) {

	int entry_pos = blockIdx.x / BLOCKS_PER_ENTRY;
	hash_entry* entry = entries + entry_pos;

	SHA256_CTX sha_ctx;
	u_hash_bytes digest;

	// int block_offset = (blockIdx.x % BLOCKS_PER_ENTRY) * (blockDim.x * LOOPS_INSIDE_THREAD);
	unsigned long long current = start + ( (blockIdx.x % BLOCKS_PER_ENTRY) * (blockDim.x * LOOPS_INSIDE_THREAD) ) + (threadIdx.x * LOOPS_INSIDE_THREAD);
	// TODO(marcosfons): Change 7 to 8 characters here
	unsigned char input[8];
	unsigned int length;

	#pragma unroll
	for (int j = 0; j < LOOPS_INSIDE_THREAD; j++) {
		length = get_input_from_number(current + j, input);

		sha256_init(&sha_ctx);
		sha256_update(&sha_ctx, entry->salt, SALT_LENGTH);
		sha256_update(&sha_ctx, input, length);
		sha256_final(&sha_ctx, digest.hash_bytes);

		if (hash_cmp_equal(digest.hash_bytes, entry->hash_bytes.hash_bytes)) {
			for (int i = 0; i < length; i++) {
				entry->solution[i] = input[i];
			}
			blockContainsSolution[entry_pos] = 1;
		}
	}
}

__global__ void sha256_cuda(hash_entry *entries, int entries_count,
                            unsigned char *blockContainsSolution,
                            unsigned long baseSeed) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
	int entry_pos = blockIdx.x % entries_count;
  unsigned long seed = deviceRandomGen(baseSeed + id);

	hash_entry* entry = entries + entry_pos;

	SHA256_CTX sha_ctx;
	u_hash_bytes digest;
	BYTE input[MAX_PASSWORD_LENGTH];

	#pragma unroll
	for (int i = 0; i < MAX_PASSWORD_LENGTH; i++) {
		seed = deviceRandomGen(seed);
		input[i] = charset[seed % CHARSET_LENGTH];
	}
	
	for (int input_length = MIN_PASSWORD_CHECK; input_length < MAX_PASSWORD_LENGTH; input_length++) {
		sha256_init(&sha_ctx);
		sha256_update(&sha_ctx, entry->salt, SALT_LENGTH);
		sha256_update(&sha_ctx, input, input_length);
		sha256_final(&sha_ctx, digest.hash_bytes);

		if (hash_cmp_equal(digest.hash_bytes, entry->hash_bytes.hash_bytes)) {
			for (int i = 0; i < input_length; i++) {
				entry->solution[i] = input[i];
			}
			blockContainsSolution[entry_pos] = 1;
		}
	}
}

void reorganize_not_solved_entries(hash_entry *entries,
                                   unsigned char *contains_solution,
                                   int entries_total, int *current_total,
                                   unsigned char *d_blockContainsSolution,
                                   hash_entry *d_hash_entry) {
	// This will place solved entries into the end of the list
	// It will change the CPU (host) variables in the launch_gpu_handler_thread function
	for (int i = 0; i < *current_total; i++) {
		if (contains_solution[i]) {
			// SWAP
			int final_index = (*current_total) - 1;
			hash_entry entry_copy = entries[i];
			unsigned char contains_solution_copy = contains_solution[i];

			entries[i] = entries[final_index];
			contains_solution[i] = contains_solution[final_index];

			entries[final_index] = entry_copy;
			contains_solution[final_index] = contains_solution_copy;

			*current_total = *current_total - 1;
		}
	}

	// Still needs to update the GPU (device) variables to reflect those
	// changes For the blockContainsSolution we can only set all to zero.
	// Because the rest will be not accessed. It sets already entries_count
	// because it easier, and it will happen not frequently
	hipMemset(d_blockContainsSolution, 0, sizeof(unsigned char) * (entries_total));
	hipMemcpy(d_hash_entry, entries, sizeof(hash_entry) * (entries_total), hipMemcpyHostToDevice);
}


unsigned char check_if_solution_was_found(unsigned char** entries_has_solution, unsigned char* d_entries_has_solution, int current_total) {
	for (int i = 0; i < current_total; i++) {
		if ((*entries_has_solution)[i]) {
			return 1;
		}
	}

	return 0;
}

void process_after_solution_was_found(hash_entry *entries, unsigned char *contains_solution,
                                   int entries_total, int *current_total,
                                   unsigned char *d_block_contains_solution,
                                   hash_entry *d_hash_entry) {
	hipMemcpy(entries, d_hash_entry, sizeof(hash_entry) * entries_total, hipMemcpyDeviceToHost);

	reorganize_not_solved_entries(
			entries, contains_solution,
			entries_total, current_total,
			d_block_contains_solution, d_hash_entry);

	printf("\n");
	for (int i = 0; i < entries_total; i++) {
		print_hash_entry(entries[i]);
	}
}


void *launch_gpu_handler_thread(void *vargp) {
	handler_input *hi = (handler_input *) vargp;
	hipSetDevice(hi->device);

	// Pre SHA-256
	hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice);

	int current_total = hi->entries_count;
	
	unsigned char *entries_has_solution = (unsigned char*) malloc(sizeof(unsigned char) * current_total);
	unsigned char *d_entries_has_solution;
	hipMalloc(&d_entries_has_solution, sizeof(unsigned char) * current_total);
	hipMemset(d_entries_has_solution, 0, sizeof(unsigned char) * current_total);

	hash_entry *d_hash_entry;
	hipMalloc(&d_hash_entry, sizeof(hash_entry) * current_total);
	hipMemcpy(d_hash_entry, hi->entries, sizeof(hash_entry) * current_total, hipMemcpyHostToDevice);


	SHA256_CTX *contexts = (SHA256_CTX*) malloc(sizeof(SHA256_CTX) * current_total);
	for (int i = 0; i < current_total; i++) {
		sha256_init(contexts + i);
		sha256_update(contexts + i, hi->entries[i].salt, SALT_LENGTH);
	}


	SHA256_CTX *d_sha256_contexts;
	hipMalloc(&d_sha256_contexts, sizeof(SHA256_CTX) * current_total);
	hipMemcpy(d_sha256_contexts, contexts, sizeof(SHA256_CTX) * current_total, hipMemcpyHostToDevice);


	#if TEST_TYPE == SEQUENTIALLY
	hi->start = 0;

	while(1) {
		for (int i = 0; i < RUNS_PER_ITERATION; i++) {
			sha256_cuda_all_posibilities<<<current_total * BLOCKS_PER_ENTRY, THREADS>>>(
					d_hash_entry, current_total, d_entries_has_solution,
					hi->start + (THREADS * i * LOOPS_INSIDE_THREAD * BLOCKS_PER_ENTRY)
			);
		}
		hipDeviceSynchronize();

		if (hi->start > 1000000000) {
			break;
		}

		hipMemcpy(entries_has_solution, d_entries_has_solution, sizeof(unsigned char) * current_total, hipMemcpyDeviceToHost);

		if (check_if_solution_was_found(&entries_has_solution, d_entries_has_solution, current_total)) {
			printf("\nSTART: %llu\n", hi->start);
			process_after_solution_was_found(
					hi->entries, entries_has_solution,
					hi->entries_count, &current_total,
					d_entries_has_solution, d_hash_entry);
		}

		hi->start += RUNS_PER_ITERATION * (THREADS * LOOPS_INSIDE_THREAD) * BLOCKS_PER_ENTRY;
		hi->hashesProcessed += RUNS_PER_ITERATION * (THREADS * LOOPS_INSIDE_THREAD) * (current_total * BLOCKS_PER_ENTRY);
	}
	hi->finished = 1;

	#elif TEST_TYPE == RANDOMLY

	srand(timems() * timems());
	while (1) {
		for (int i = 0; i < RUNS_PER_ITERATION; i++) {
			sha256_cuda<<<THREADS, BLOCKS>>>(d_hash_entry, current_total, d_entries_has_solution, rand());
		}
		hipDeviceSynchronize();
		hi->hashesProcessed += THREADS * BLOCKS * THREAD_EXECUTION_ITERATIONS * RUNS_PER_ITERATION;
			
		hipMemcpy(entries_has_solution, d_entries_has_solution, sizeof(unsigned char) * current_total, hipMemcpyDeviceToHost);

		if (check_if_solution_was_found(&entries_has_solution, d_entries_has_solution, current_total)) {
			process_after_solution_was_found(
					hi->entries, entries_has_solution,
					hi->entries_count, &current_total,
					d_entries_has_solution, d_hash_entry);
		}
	}

	#endif

  hipDeviceReset();
  return NULL;
}


int main() {
	setlocale(LC_NUMERIC, "");

	show_devices_info();

	int entries_count = 0;
	hash_entry* entries = (hash_entry*) malloc(1);
	printf("Loading hashes from the file\n\n");

	read_entries_from_file("data/hashes_and_salts.txt", &entries, &entries_count);
	// read_entries_from_file("data/correct.txt", &entries, &entries_count);
	if (entries_count == 0) {
		printf("No entries found, exiting\n");
		exit(0);
	}

	for (int i = 0; i < entries_count; i++) {
		print_hash_entry(entries[i]);
	}

	printf("\nStarting to break hashes\n");


	// 6f416ce900e7a39206334a28b40f609a2984332b2b5313cdafba10e2f3d6f3a5:HFq..h :abcDef
	// 94d72fe5153921c8b5ccee30e639025c7640ad15ed4c2c68e1eacb6d2db94139:G3m"5,N:1@2@3@4@
	// 0a6ab9b4100383117271cd5c7ce083be7bbb669a532cc8857356315e61340abe:*3~/]cXER:passworD
	// a775bf388c6e99f7255169afa0769b594692d86c662f294057de91a182cb416f:]x<7aV,1:p@ssw0rd
	// 66240965684bed7ecd3ec495208364f25e964fe83aa31679f3210a5bfe32dc10:E3U:12081786
	// c6f415b777999c168533a0a2716e6125f740235e99c03319ef0dcb1a0be06c15:?/ثFz9g馿f:00000000
	// 69017d19f71e8e34d5a53be54ca8d4d7bc9dc6c913babe3bb1e222010eba8066:t\|p,:AbCdEfGh
	// 12fad8a9aeb1c8ed1f988b07b32f0a9b7d7458e7c99822d1d4284bf6edcf3a3e:; 5W/g:M3t@llic@
	// 27a575da417e1e4cdbf4fbbe8752579b6e1d65e79731ed773a6886812e2da116:3Tb:,b$]:6%Fg

	unsigned long long **processedPtrs = (unsigned long long **) malloc(sizeof(unsigned long long *) * GPUS);
	unsigned long long **singleProcessedPtrs = (unsigned long long **) malloc(sizeof(unsigned long long *) * GPUS);
	unsigned char **finished_ptrs = (unsigned char**) malloc(sizeof(unsigned char*) * GPUS);

	pthread_t *tids = (pthread_t *) malloc(sizeof(pthread_t) * GPUS);
	unsigned long long start = timems();
	for (int i = 0; i < GPUS; i++) {
		handler_input *hi = (handler_input*) malloc(sizeof(handler_input));
		hi->device = i;
		hi->hashesProcessed = 0;
		hi->entries = entries;
		hi->entries_count = entries_count;
		hi->finished = 0;
		processedPtrs[i] = &hi->hashesProcessed;
		singleProcessedPtrs[i] = &hi->start;
		finished_ptrs[i] = &hi->finished;
		pthread_create(tids + i, NULL, launch_gpu_handler_thread, hi);
	}

	while (1) {
		usleep(PRINT_STATUS_DELAY);
		unsigned long totalProcessed = 0;
		unsigned long singleProcesseds = 0;
		unsigned char finished = 1;
		for (int i = 0; i < GPUS; i++) {
			singleProcesseds = *(singleProcessedPtrs[i]);
			totalProcessed += *(processedPtrs[i]);
			finished = finished && *(finished_ptrs[i]);
		}

		long long elapsed = timems() - start;
		printf("Total Hashes (%'lu) Hashes per Salt (%'lu) Seconds (%'f) Hashes/sec (%'lu)     \r",
					totalProcessed, 
				 singleProcesseds,
				 ((float) elapsed) / 1000.0,
					(unsigned long) ((double) totalProcessed / (double) elapsed) * 1000);

		if (finished) {
			break;
		}
	}

	long long elapsed = timems() - start;

	for (int i = 0; i < GPUS; i++) {
		pthread_join(tids[i], NULL);
	}

	unsigned long totalProcessed = 0;
	for (int i = 0; i < GPUS; i++) {
		totalProcessed += *(processedPtrs[i]);
	}

	printf("\nHashes processed: %'lu\n", totalProcessed);
	printf("Time: %llu\n", elapsed);
	printf("Hashes/sec: %'lu\n", (unsigned long) ((double) totalProcessed / (double) elapsed) * 1000);

	free(entries);

	return 0;
}

