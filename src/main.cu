#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <pthread.h>
#include <stdlib.h>
#include <unistd.h>
#include <locale.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "sha256.cuh"
#include "wordlist.h"
#include "hash_entry.h"
#include "cuda_devices.h"


// #define TEST_TYPE SEQUENTIAL_WORDLIST
#define TEST_TYPE SEQUENTIALLY
// #define TEST_TYPE RANDOMLY

#define BLOCKS_PER_ENTRY 300
#define THREADS 1024
#define RUNS_PER_ITERATION 1
#define LOOPS_INSIDE_THREAD 66
#define PRINT_STATUS_DELAY 10000

#define THREAD_EXECUTION_ITERATIONS ((MAX_PASSWORD_LENGTH - MIN_PASSWORD_CHECK))

#define MAX_SEQUENTIAL_WORDLIST_CHARS 600000000

const char *CHARSET = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ1234567890%*$@";
// const char CHARSET[] = {'1', '2', '3', '4', '5', '6', '7', '8', '9', '0'};

typedef struct HandlerInput {
	unsigned long long hashesProcessed;
	unsigned long long start;
	HashEntries entries;
	bool finished;
	SequentialWordlist *sequentialWordlist;
} HandlerInput;

long long timems() {
	struct timeval end;
	gettimeofday(&end, NULL);
	return (long long) end.tv_sec * 1000 + (long long) end.tv_usec / 1000;
}

__forceinline__ __device__ bool hashCompare(const u_HashBytes *hash1,
                            const u_HashBytes *hash2) {
	#pragma unroll
	for (unsigned short i = 0; i < HASH_BYTES_LENGTH; i++) {
		if (hash1->bytes[i] != hash2->bytes[i]) {
			return 0;
		}
	}
	return 1;
}

__global__ void sha256SequentialWordlist(HashEntry *__restrict__ entries,
                                         int entriesCount,
                                         unsigned long long start,
                                         unsigned char *sequentialWordlist) {
	HashEntry* entry = entries + blockIdx.x;

	SHA256_CTX shaCtx;
	u_HashBytes digest;
	unsigned long long step = start + (blockIdx.y * THREADS) + threadIdx.x;

	#pragma unroll
	for (unsigned short j = MIN_PASSWORD_CHECK; j < MAX_PASSWORD_LENGTH + 1; j++) {
		sha256_init(&shaCtx);
		#pragma unroll
		for (unsigned short i = 0; i < SALT_LENGTH; i++) {
			shaCtx.data[i] = entry->salt[i];
		}
		shaCtx.datalen = SALT_LENGTH;

		sha256_update(&shaCtx, sequentialWordlist + step, j);
		sha256_final(&shaCtx, digest.bytes);

		if (hashCompare(&digest, &entry->hashBytes)) {
			for (int i = 0; i < j; i++) {
				entry->solution[i] = sequentialWordlist[step + i];
			}
			if (j < MAX_PASSWORD_LENGTH) {
				entry->solution[j] = '\0';
			}
			return;
		}
	}
}

void processAfterSolutionWasFound(HashEntries *entries,
                                  HashEntry *d_hashEntry) {
	reorganizeNotSolvedEntries(entries);
	hipMemcpy(d_hashEntry, entries->entries, sizeof(HashEntry) * (entries->entriesCount), hipMemcpyHostToDevice);

	printf("\n");
	printHashEntries(entries);
}

void *launchGPUHandlerThread(void *vargp) {
	HandlerInput *hi = (HandlerInput *) vargp;

	// Pre SHA-256
	hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice);
	
	HashEntry *d_hashEntry;
	hipMalloc(&d_hashEntry, sizeof(HashEntry) * hi->entries.entriesCount);
	hipMemcpy(d_hashEntry, hi->entries.entries, sizeof(HashEntry) * hi->entries.entriesCount, hipMemcpyHostToDevice);

	unsigned char* d_wordlist;
	hipMalloc(&d_wordlist, sizeof(unsigned char) * hi->sequentialWordlist->characterCount);
	hipMemcpy(d_wordlist, hi->sequentialWordlist->words, sizeof(unsigned char) * hi->sequentialWordlist->characterCount, hipMemcpyHostToDevice);
	hi->sequentialWordlist->copied = true;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hi->start = 0;

	while(1) {
		dim3 numBlocks(hi->entries.currentTotal, BLOCKS_PER_ENTRY, 1);
		dim3 numThreads(THREADS, 1, 1);

		hipEventRecord(start);
		sha256SequentialWordlist<<<numBlocks, numThreads>>>(
				d_hashEntry, hi->entries.currentTotal, hi->start, d_wordlist
		);
		hipEventRecord(stop);
		hi->start += BLOCKS_PER_ENTRY * THREADS;
		// hipDeviceSynchronize();
		hipEventSynchronize(stop);

		// float milliseconds = 0;
		// hipEventElapsedTime(&milliseconds, start, stop);

		int processed = hi->entries.currentTotal * (MAX_PASSWORD_LENGTH - MIN_PASSWORD_CHECK) * THREADS * BLOCKS_PER_ENTRY;
		// printf("Milliseconds elapsed: %f  Milliseconds per execution %f  Processed: %d\r", milliseconds, milliseconds / (processed), processed);

		hipMemcpy(hi->entries.entries, d_hashEntry, sizeof(HashEntry) * hi->entries.currentTotal, hipMemcpyDeviceToHost);

		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			printf("Error cuda sha256 hash: %s \n", hipGetErrorString(error));
		}

		if (containsNewSolution(&hi->entries)) {
			processAfterSolutionWasFound(&hi->entries, d_hashEntry);
		}

		hi->hashesProcessed += hi->entries.currentTotal * (MAX_PASSWORD_LENGTH - MIN_PASSWORD_CHECK) * THREADS * BLOCKS_PER_ENTRY;

		if (hi->start > hi->sequentialWordlist->characterCount) {
			hi->start = 0;
			hipMemcpy(d_wordlist, hi->sequentialWordlist->words, sizeof(unsigned char) * hi->sequentialWordlist->characterCount, hipMemcpyHostToDevice);
			printf("\nCopied to GPU\n");
			hi->sequentialWordlist->copied = true;
			if (hi->sequentialWordlist->finished) {
				break;
			}
		}
	}
	hi->finished = true;


  hipDeviceReset();
  return NULL;
}


int main(int argc, char *argv[]) {
	setlocale(LC_NUMERIC, "");

	SequentialWordlist wordlist;
	// createSequentialWordlist(&wordlist, 9, CHARSET, sizeof(CHARSET), MAX_SEQUENTIAL_WORDLIST_CHARS);

	// for (size_t i = 0; i < wordlist.words_count; i++) {
	// 	for (int j = 0; j < wordlist.word_length; j++) {
	// 		printf("%c", wordlist.words[(i * wordlist.word_length) + j]);
	// 	}
	// 	printf("\n");
	// }

	showGPUDevicesInfo();

	srand(time(NULL));

	


	// createSequentialWordlistFromFile(&wordlist, "/mnt/wordlist/all_in_one_p", 1500000000);
	createSequentialWordlistFromFile(&wordlist, "/mnt/wordlist/all_in_one_p", 500000000);
	// createSequentialWordlistFromFile(&wordlist, "/mnt/wordlist/weakpass_3a", 1500000000);
	// createSequentialWordlistFromFile(&wordlist, "wordlists/new_shuf.txt", 10000000);
	// createSequentialWordlistFromFile(&wordlist, "/mnt/wordlist/weakpass_3a", 300000);
	// createSequentialWordlistFromFile(&wordlist, "wordlists/n_crackstation-human-only.txt", 300);
	// createSequentialWordlistFromFile(&wordlist, "wordlists/passwords.txt", 10000000);

	readNextChunkFromSequentialWordlist(&wordlist, CHARSET);

	// readSequentialWordlistFromFile("/mnt/wordlist/weakpass_3a", &wordlist, CHARSET);
	// readSequentialWordlistFromFile("wordlists/n_crackstation-human-only.txt", &wordlist, CHARSET);
	// readSequentialWordlistFromFile("wordlists/n_crackstation-human-only.txt", &wordlist);

	printf("Loading hashes from the file\n\n");
	HashEntries entries;
	readHashEntriesFromFile("data/hashes_and_salts.txt", &entries);
	if (entries.entriesCount == 0) {
		printf("No entries found, exiting\n");
		exit(0);
	}

	printHashEntries(&entries);

	printf("\nStarting to break hashes\n");

	pthread_t threadId;
	unsigned long long start = timems();

	HandlerInput input;
	input.hashesProcessed = 0;
	input.entries = entries;
	input.finished = false;
	input.sequentialWordlist = &wordlist;

	pthread_create(&threadId, NULL, launchGPUHandlerThread, &input);

	while (1) {
		usleep(PRINT_STATUS_DELAY);
		unsigned long long singleProcesseds = input.start;
		unsigned long long totalProcessed = input.hashesProcessed;

		long long elapsed = timems() - start;
		printf("Total Hashes (%'llu) Hashes per Salt (%'llu) Seconds (%'f) Hashes/sec (%'llu)\r",
					totalProcessed, singleProcesseds, ((double) elapsed) / 1000.0,
					(unsigned long long) ((double) totalProcessed / (double) elapsed) * 1000
		);

		if (wordlist.copied) {
			printf("\nGetting more passwords\n");
			readNextChunkFromSequentialWordlist(&wordlist, CHARSET);
			printf("Got more passwords\n");
		}

		if (input.finished) {
			break;
		}
	}

	long long elapsed = timems() - start;

	pthread_join(threadId, NULL);

	printf("\nHashes processed: %'llu\n", input.hashesProcessed);
	printf("Time: %llu\n", elapsed);
	printf("Hashes/sec: %'lu\n", (unsigned long) ((double) input.hashesProcessed / (double) elapsed) * 1000);

	return 0;
}

