#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <pthread.h>
#include <locale.h>

#include "sha256.cuh"
#include "hash_entry.cuh"


#define THREADS 1500
// #define THREADS 1500
#define BLOCKS 256
// #define BLOCKS 256
#define GPUS 1

#define THREAD_EXECUTION_ITERATIONS 20

#define CHARSET_LENGTH 66
__constant__ BYTE charset[CHARSET_LENGTH + 1] = {"abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ1234567890%*$@"};

char* g_solution;

__host__ __device__ void print_hash_entry(hash_entry entry) {
	printf("Hash: ");
	for (size_t i = 0; i < HASH_BYTES_LENGTH; i++) {
		printf("%02x", entry.hash_bytes[i]);
	}
	printf("\nSalt: %.32s\n", entry.salt);
}

__device__ unsigned long deviceRandomGen(unsigned long x) {
  x ^= (x << 21);
  x ^= (x >> 35);
  x ^= (x << 4);
  return x;
}

__global__ void sha256_cuda(hash_entry *entry, int *blockContainsSolution, unsigned long baseSeed) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long seed = deviceRandomGen(baseSeed + id);

	int password_size = (seed % MAX_PASSWORD_LENGTH - 1) + 2;

	BYTE input[SALT_LENGTH + MAX_PASSWORD_LENGTH];
	memcpy(input, entry->salt, SALT_LENGTH);

	SHA256_CTX sha_ctx;
	BYTE digest[32];

	int found;

	for (int i = 0; i < password_size; i++) {
		seed = deviceRandomGen(seed);
		input[SALT_LENGTH + i] = charset[seed % CHARSET_LENGTH];
	}

	for (int x = 0; x < THREAD_EXECUTION_ITERATIONS; x++) {
		seed = deviceRandomGen(seed);
		input[SALT_LENGTH + (seed % password_size)] = charset[seed % CHARSET_LENGTH];
		seed = deviceRandomGen(seed);
		input[SALT_LENGTH + (seed % password_size)] = charset[seed % CHARSET_LENGTH];

		sha256_init(&sha_ctx);
		// sha256_update(&sha_ctx, input, (SALT_LENGTH);
		sha256_update(&sha_ctx, input, (SALT_LENGTH + password_size));
		sha256_final(&sha_ctx, digest);

		found = 1;
		for (int i = 0; i < HASH_BYTES_LENGTH; i++) {
			if (digest[i] != entry->hash_bytes[i]) {
				found = 0;
				break;
			}
		}

		if (found) {
			break;
		}
	}

	if (found) {
		for (int i = 0; i < password_size; i++) {
			entry->solution[i] = input[SALT_LENGTH + i];
		}
		for (int i = password_size; i < MAX_PASSWORD_LENGTH; i++) {
			entry->solution[i] = '\0';
		}
		*blockContainsSolution = 1;
	}
}

void hostRandomGen(unsigned long *x) {
  *x ^= (*x << 21);
  *x ^= (*x >> 35);
  *x ^= (*x << 4);
}

void pre_sha256() {
  hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice);
}

long long timems() {
  struct timeval end;
  gettimeofday(&end, NULL);
  return end.tv_sec * 1000LL + end.tv_usec / 1000;
}

struct HandlerInput {
  int device;
  unsigned long hashesProcessed;
	hash_entry entry;
};
typedef struct HandlerInput HandlerInput;

pthread_mutex_t solutionLock;

void *launchGPUHandlerThread(void *vargp) {
  HandlerInput *hi = (HandlerInput *) vargp;
  hipSetDevice(hi->device);

  pre_sha256();

	hash_entry *d_hash_entry;
	hipMalloc(&d_hash_entry, sizeof(hash_entry));
	hipMemcpy(d_hash_entry, &(hi->entry), sizeof(hash_entry), hipMemcpyHostToDevice);

  int blockContainsSolution = 0;
  int *d_blockContainsSolution;
  hipMalloc(&d_blockContainsSolution, sizeof(int));
	hipMemcpy(&blockContainsSolution, d_blockContainsSolution, sizeof(int), hipMemcpyHostToDevice);

  unsigned long rngSeed = timems();

  // while (1) {
	srand(rngSeed);
  for (int i = 0; i < 30000; i++) {
		rngSeed = rand();

    hi->hashesProcessed += THREADS * BLOCKS * THREAD_EXECUTION_ITERATIONS;
    sha256_cuda<<<THREADS, BLOCKS>>>(d_hash_entry, d_blockContainsSolution, rngSeed);
    hipDeviceSynchronize();

    hipMemcpy(&blockContainsSolution, d_blockContainsSolution, sizeof(int), hipMemcpyDeviceToHost);

		if (blockContainsSolution == 1) {
			char* solution = (char*) malloc(sizeof(char) * MAX_PASSWORD_LENGTH);
      hipMemcpy(solution, &(d_hash_entry->solution), sizeof(char) * MAX_PASSWORD_LENGTH, hipMemcpyDeviceToHost);
			printf("\nSolution: %s\n", solution);
			exit(1);
			break;
		}
  }

  hipDeviceReset();
  return NULL;
}

void hexToBytes(const char* hex_string, BYTE bytes[HASH_BYTES_LENGTH]) {
  for (unsigned int i = 0; i < HASH_BYTES_LENGTH; i += 1) {
		sscanf(&hex_string[i * 2], "%02x", (unsigned int *) &bytes[i]);
  }
}


int main() {
	setlocale(LC_NUMERIC, "");

	hash_entry line;
	// hexToBytes("27a575da417e1e4cdbf4fbbe8752579b6e1d65e79731ed773a6886812e2da116", line.hash_bytes);
	// strncpy(line.salt, "3354623a2c1deaed1362f124c75db8a7", SALT_LENGTH);
	hexToBytes("6cea8869d44eefacc4b56d300905b9aa770503b46cca36f7cec9b36c8bb45ded", line.hash_bytes);
	strncpy(line.salt, "2609ad21084c3cc3e64f0e6777466000", SALT_LENGTH);
	print_hash_entry(line);

	pthread_mutex_init(&solutionLock, NULL);
	pthread_mutex_lock(&solutionLock);

	unsigned long **processedPtrs = (unsigned long **) malloc(sizeof(unsigned long *) * GPUS);
	pthread_t *tids = (pthread_t *) malloc(sizeof(pthread_t) * GPUS);
	long long start = timems();
	for (int i = 0; i < GPUS; i++) {
    HandlerInput *hi = (HandlerInput *) malloc(sizeof(HandlerInput));
    hi->device = i;
    hi->hashesProcessed = 0;
		hi->entry = line;
    processedPtrs[i] = &hi->hashesProcessed;
    pthread_create(tids + i, NULL, launchGPUHandlerThread, hi);
    usleep(10);
	}

	// while (1) {
	usleep(100000);
	for (int i = 0; i < 300000; i++) {
		usleep(1000);
		unsigned long totalProcessed = 0;
		for (int i = 0; i < GPUS; i++) {
			totalProcessed += *(processedPtrs[i]);
		}
		long long elapsed = timems() - start;
		printf("Hashes (%'lu) Seconds (%'f) Hashes/sec (%'lu)\r", totalProcessed, ((float) elapsed) / 1000.0, (unsigned long) ((double) totalProcessed / (double) elapsed) * 1000);
		if (g_solution) {
			break;
		}
	}
	printf("\n");

	pthread_mutex_lock(&solutionLock);
	long long end = timems();
	long long elapsed = end - start;

	for (int i = 0; i < GPUS; i++) {
		pthread_join(tids[i], NULL);
	}

	unsigned long totalProcessed = 0;
	for (int i = 0; i < GPUS; i++) {
		totalProcessed += *(processedPtrs[i]);
	}

	printf("Solution: %s\n", g_solution);
	printf("Hashes processed: %'lu\n", totalProcessed);
	printf("Time: %llu\n", elapsed);
	printf("Hashes/sec: %'lu\n", (unsigned long) ((double) totalProcessed / (double) elapsed) * 1000);

	return 0;
}
